﻿#define STB_IMAGE_IMPLEMENTATION 
#include "C:\\Users\\andre\\Documents\\COMPUTACION_PARALELA\\Librerias\\stb_image.h"

#include <iostream>
#include <cstdlib>
#include <cmath>

int main() {
    // Rutas fijas de tus imágenes a comparar
    const char* path_cpu = "C:\\Users\\andre\\Documents\\61CPU.jpg";
    const char* path_gpu = "C:\\Users\\andre\\Documents\\61Cuda.jpg";

    int w1, h1, ch1;
    int w2, h2, ch2;

    // Cargar imagen procesada por CPU
    unsigned char* img1 = stbi_load(path_cpu, &w1, &h1, &ch1, 3);
    if (!img1) {
        std::cerr << "Error cargando imagen CPU: " << path_cpu << "\n";
        return 1;
    }

    // Cargar imagen procesada por GPU
    unsigned char* img2 = stbi_load(path_gpu, &w2, &h2, &ch2, 3);
    if (!img2) {
        std::cerr << "Error cargando imagen GPU: " << path_gpu << "\n";
        stbi_image_free(img1); // Liberar la imagen anterior si falla esta
        return 1;
    }

    // Verificar que ambas imágenes tengan las mismas dimensiones
    if (w1 != w2 || h1 != h2) {
        std::cerr << "Las imágenes tienen diferentes dimensiones\n";
        stbi_image_free(img1);
        stbi_image_free(img2);
        return 1;
    }

    // Calcular el número total de píxeles
    int total_pixels = w1 * h1;
    int diff_pixels = 0;

    // Recorrer todos los píxeles (3 canales por píxel: R, G, B)
    for (int i = 0; i < total_pixels * 3; i += 3) {
        int r1 = img1[i], g1 = img1[i + 1], b1 = img1[i + 2];
        int r2 = img2[i], g2 = img2[i + 1], b2 = img2[i + 2];

        // Contar como diferente si al menos un canal es distinto
        if (r1 != r2 || g1 != g2 || b1 != b2) {
            diff_pixels++;
        }
    }

    // Calcular porcentaje de diferencia e igualdad
    float porcentaje_diferencia = (100.0f * diff_pixels) / total_pixels;
    float porcentaje_igualdad = 100.0f - porcentaje_diferencia;

    // Mostrar resultados según coincidencias
    if (diff_pixels == 0) {
        std::cout << "Las imágenes son completamente iguales.\n";
    }
    else {
        std::cout << "Las imágenes son diferentes.\n";
    }

    std::cout << "Porcentaje de igualdad: " << porcentaje_igualdad << "%\n";

    // Liberar memoria de imágenes cargadas
    stbi_image_free(img1);
    stbi_image_free(img2);

    return 0;
}
